#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
 
__global__ void kernel(int *a) 
{
  int index = threadIdx.x;
  a[index] *= 2;
  printf("%d\t", a[index]);
}
 
void printVector(int *in, int n)
{
 printf("\nThis is the host\n");
 for(int i = 0; i < n; i++)
  printf("%d\t", in[i]);
 printf("\n");
}

int main(int argc, char* argv[]) 
{
  /*Variables*/
  int dataSize = 8;
  int nGPUs = 0;
  hipGetDeviceCount(&nGPUs); 
  printf("nGPUs = %d\n",nGPUs);
    
  int *h_data       = (int*)  malloc (dataSize  * sizeof(int));
  int **d_data      = (int**) malloc (nGPUs     * sizeof(int*));
  
  int *DeviceList   = (int *) malloc (nGPUs     * sizeof(int));
  for(int i = 0; i < nGPUs; i++)
      DeviceList[i] = i;
  
  /*Initializing NCCL with Multiples Devices per Thread*/
  ncclComm_t* comms      = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * nGPUs);  
  hipStream_t* stream   = (hipStream_t*)malloc(sizeof(hipStream_t)* nGPUs);
  ncclCommInitAll(comms, nGPUs, DeviceList);
  
  /*Population the data vector*/
  for(int i = 0; i < dataSize; i++)
      h_data[i] = rand()%(10-2)*2;
 
  printVector(h_data, dataSize);
      
  for(int g = 0; g < nGPUs; g++) 
  {
      hipSetDevice(g);
      hipStreamCreate(&stream[g]);
      hipMalloc(&d_data[g], dataSize * sizeof(int));
     
      if(g == 0)  /*Copy from Host to Device*/
         hipMemcpy(d_data[g], h_data, dataSize * sizeof(int), hipMemcpyHostToDevice);
  }
        
  ncclGroupStart();
  for(int g = 0; g < nGPUs; g++) 
  {
    hipSetDevice(DeviceList[g]);
    ncclBcast(d_data[g], dataSize, ncclInt, 0, comms[g], stream[g]); /*Broadcasting it to all*/
  }
  ncclGroupEnd();       

  for(int g = 0; g < nGPUs; g++) 
  {
    hipSetDevice(DeviceList[g]);
    printf("\nThis is the device [%d]\n", g);
    kernel <<< 1 , dataSize >>> (d_data[g]); /*Call the CUDA Kernel*/
    hipDeviceSynchronize();             
  }

  printf("\n");
    
  for(int g = 0; g < nGPUs; g++)  /*Synchronizing CUDA Streams*/
  {                                
    hipSetDevice(DeviceList[g]);
    hipStreamSynchronize(stream[g]);
  }
 
  for(int g = 0; g < nGPUs; g++)   /*Destroy CUDA Streams*/
  {                                
    hipSetDevice(DeviceList[g]);
    hipStreamDestroy(stream[g]);
  }

  for(int g = 0; g < nGPUs; g++)   /*Finalizing NCCL*/
     ncclCommDestroy(comms[g]);
  
  /*Freeing memory*/
  free(h_data); 
  free(DeviceList);
  hipFree(stream);
  hipFree(d_data);

  return 0;
}
